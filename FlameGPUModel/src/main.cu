#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <filesystem>
#include <mutex>
#include <vector>   

#include "flamegpu/flamegpu.h"

#include <GL/glew.h>
#include <GL/GL.h>
#include <GL/GLU.h>
#include <GL/freeglut.h>
#include <GL/glut.h>

// Grid Size ����������� ����������� ������������ � ��������
#define GRID_WIDTH 800
#define GRID_HEIGHT 800
#define dim 100 // ����������� ������������ 1000 x 1000
#define side_size 8 // ������ ������� ��������
#define TIME_STOP 80 // ���������� �������� ������� (���)
#define RUN_COUNT 100 // ���������� �������� ��� ��������
#define NumberOfCitizens 1000
#define NumberOfMigrants 100

// Visualisation mode (1=standalone run, 0 = essemble run)
#define VIS_MODE 1


std::mutex m;

int window_width = 1050;
int window_height = 1050;

int window_id = 0;

float x_a[100000], y_a[100000]; //���������� �������
int agent_type[100000]; // ��� �������
int a_size = 0; // ���������� �������


std::vector <double> x_cell; // ���������� ���� ����� � ����������� ������������
std::vector <double> y_cell; // 

uint8_t resource_type[dim][dim]; // ��� �������

std::ofstream out("results.txt", std::ios::app);
std::ofstream out2("log.txt", std::ios::app); // ��� ��� �������� �������


std::atomic<bool> occupied_cells[dim][dim] = { }; // ������� ��� ��������� ������ ���������� ������������

void display(void);

extern void initVisualisation();
extern void runVisualisation();

__shared__ unsigned int agent_nextID;

__host__ __device__  unsigned int getNextID() {
    agent_nextID++;
    return agent_nextID;
}

//������� ���������� ������ ������
__host__ __device__ double kids_p(unsigned int kids, unsigned int current_age)
{
double w = 0.0;
if (kids > 0 && current_age > 0)
w = (double)exp(-1 / (pow(kids, 0.9) * pow(current_age, 0.1)));
return w;
}

class Cells {
public:
  //  std::mutex mtx;
    float x;
    float y;
    int r;
    int c;
    int ClusterIndex = -1; // ������ �������� ������
    
Cells (float x, float y, int r, int c) {
     //std::lock_guard<std::mutex> lock(mtx);
       this->x = x;
       this->y = y;
       this->r = r;
       this->c = c;
       ClusterIndex = -1;
    }
};


// ������ ������� ���������� �������
__host__ double Dunkan_segregation_index(int locals_count, int foreigns_count, std::vector< std::vector<int> >& agent_in_cell)
{
    double DSI = 0;
    int count_1 = 0;
    int count_2 = 0;
    int i_start = 0;
    int j_start = 0;
    int vicinity_size = 33; // ����������� ����������� = (int) dim (100) / 3

    for (int v_j = 0; v_j < vicinity_size; v_j++)
    {
        i_start = 0;
        for (int v_i = 0; v_i < vicinity_size; v_i++)
        {
            count_1 = 0;
            count_2 = 0;

            for (int j = j_start; j < j_start + 3; j++)
            {
                for (int i = i_start; i < i_start + 3; i++)
                {
                    if (agent_in_cell[i][j] == 1)
                        count_1++;
                    if (agent_in_cell[i][j] == 2)
                        count_2++;
                }
            }

            if (locals_count > 0 && foreigns_count > 0)
                DSI = DSI + 0.5 * abs((double)count_1 / locals_count - (double)count_2 / foreigns_count);

            i_start = i_start + 3;
        }
        j_start = j_start + 3;
    }

    if (locals_count == 0 || foreigns_count == 0)
        DSI = 1;

    return DSI;
}


//********************************* ������ ��� ������������� ������� (������� ����) *******************************************
class ClusterCentroid {
    // ����� ��������
private:
    double X;
    double Y;
public:
    double getClusterX() { return X; };
    double getClusterY() { return Y; };
    void setClusterX(double x) { this->X = x; };
    void setClusterY(double y) { this->Y = y; };

    ClusterCentroid(Cells p)
    {
        this->X = p.x;
        this->Y = p.y;
    }
};

// ClusterPoint
class ClusterPoint {
    // ���������� ������ ��������
private:
    double X;
    double Y;
    int ClusterIndex;

    /// Gets or sets X-coord of the point
    /// 
public:
    double getX() { return X; }
    double getY() { return Y; }
    void setX(double x) { this->X = x; }
    void setY(double y) { this->Y = y; }

    ClusterPoint(Cells p)
    {
        this->X = p.x;
        this->Y = p.y;
        this->ClusterIndex = -1;
    }
};


//CMeansAlgorithm
class CMeansAlgorithm {

    /// Array containing all points-agents used by the algorithm
private:
    std::vector<Cells> Points;
    /// Array containing all clusters-agents handled by the algorithm
    std::vector<Cells> Clusters;
    std::vector<Cells> centrs;
    double Fuzzyness;
    /// Algorithm precision
    double Eps = pow(10, -5);
    double getJ() { return J; }
    void setJ(double j) { this->J = j; }


    /// Recalculates cluster indexes
    void RecalculateClusterIndexes(std::vector<Cells>& Points)
    {
        for (int i = 0; i < Points.size(); i++)
        {
            double max = -1.0;

            for (int j = 0; j < Clusters.size(); j++)
            {
                if (max < U[i][j]) // && U[i][j] >= 0.6
                {
                    max = U[i][j];
                    Points[i].ClusterIndex = j;
                }
            }
        }
    }
    /// CalculateObjectiveFunction
    double CalculateObjectiveFunction(std::vector<Cells>& Points, int max_rank, double max_distance)
    {
        double Jk = 0;

        for (int i = 0; i < Points.size(); i++)
        {
            for (int j = 0; j < Clusters.size(); j++)
            {
                Jk += pow(U[i][j], Fuzzyness) * (pow(Points[i].x - centrs[j].x, 2) + pow(Points[i].y - centrs[j].y, 2));
            }
        }
        return Jk;
    }

/// Calculates the centroids of the clusters 
    void CalculateClusterCenters(std::vector<Cells>& Points, int max_rank, double max_distance)
    {
        for (int j = 0; j < Clusters.size(); j++)
        {
            double uX = 0.0;
            double uY = 0.0;
            double l = 0.0;

            for (int i = 0; i < Points.size(); i++)
            {
                ClusterPoint p(Points[i]);
                double uu = pow(U[i][j], Fuzzyness);

                uX += uu * p.getX();
                uY += uu * p.getY();
                l += uu;
            }
            
            centrs[j].x = uX / l;
            centrs[j].y = uY / l;
        }
    }



public:
    std::vector< std::vector<double> > U;

    double J; // Minimizing criterion

    double getCentersX(int ClusterIndex)
    {
        double c_x = 0.0;
        c_x = centrs.at(ClusterIndex).x;
        return c_x;
    }

    double getCentersY(int ClusterIndex)
    {
        double c_y = 0.0;
        c_y = centrs.at(ClusterIndex).y;
        return c_y;
    }


/// 
/// Perform one step of the algorithm
/// 
    void Step(std::vector<Cells>& Points, int max_rank, double max_distance)
    {
        double diff;
        for (int i = 0; i < Points.size(); i++)
        {
            ClusterPoint p(Points[i]);
            double sum = 0.0;
            for (int j = 0; j < Clusters.size(); j++)
            {
                diff = sqrt(pow(p.getX() - centrs[j].x, 2.0) + pow(p.getY() - centrs[j].y, 2.0));
                diff = (diff == 0) ? Eps : diff;
                U[i][j] = 1.0 / pow(diff, 2.0 / (Fuzzyness - 1.0));
                sum += U[i][j];
            }

            for (int j = 0; j < Clusters.size(); j++)
            {
                U[i][j] = U[i][j] / sum;
            }
        }

        RecalculateClusterIndexes(Points);
    }

    /// 
    /// Perform a complete run of the algorithm until the desired accuracy is achieved.
    /// For demonstration issues, the maximum Iteration counter is set to 20.
    /// 
    /// Algorithm accuracy
    /// The number of steps the algorithm needed to complete
    int Run(double accuracy, std::vector<Cells>& Points, int max_rank, double max_distance)
    {
        int i = 0;
        int maxIterations = 100;
        do
        {
            i++;
            J = CalculateObjectiveFunction(Points, max_rank, max_distance);
            CalculateClusterCenters(Points, max_rank, max_distance);
            Step(Points, max_rank, max_distance);
            double Jnew = CalculateObjectiveFunction(Points, max_rank, max_distance);
            if (abs(J - Jnew) < accuracy) break;
        } while (maxIterations > i);
        return i;
    }


    CMeansAlgorithm(std::vector<Cells>& Points, std::vector<Cells>& clusters, float fuzzy) {

        this->Clusters = clusters;

         U.assign(Points.size(), std::vector<double>(this->Clusters.size()));
       // U.resize(Points.size());

        this->Fuzzyness = fuzzy;

        double diff;

        // Iterate through all points to create initial U matrix
        for (int i = 0; i < Points.size(); i++)
        {
            ClusterPoint p(Points[i]);
            double sum = 0.0;

            for (int j = 0; j < Clusters.size(); j++)
            {
                centrs.push_back(Clusters[j]);
                diff = sqrt(pow(p.getX() - centrs[j].x, 2.0) + pow(p.getY() - centrs[j].y, 2.0));
                diff = (diff == 0) ? Eps : diff;
                U[i][j] = 1.0 / pow(diff, 2.0 / (Fuzzyness - 1.0));
                sum += U[i][j];
            }

            for (int j = 0; j < this->Clusters.size(); j++)
            {
                U[i][j] = U[i][j] / sum;
            }
        }
        RecalculateClusterIndexes(Points);
    }
};
//************************************************************************************************************************
int initGL()
{
    // initialize necessary OpenGL extensions
    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 "
        "GL_ARB_pixel_buffer_object"
    )) {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return FALSE;
    }
    // default initialization
    glClearColor(1.0, 1.0, 1.0, 1.0);
}

void reshape(int w, int h)
{
    glViewport(0, 0, w, h);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, w, 0, h);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

void windowResize(int width, int height) {
    window_width = width;
    window_height = height;
}

extern void initVisualisation()
{
    // Create GL context
    int   argc = 1;
    char glutString[] = "GLUT application";
    char* argv[] = { glutString, NULL };
    //char *argv[] = {"GLUT application", NULL};	

    glutInit(&argc, argv);


    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    window_id = glutCreateWindow("FLAME GPU Visualiser");
    glutReshapeFunc(windowResize);

    // initialize GL
    if (FALSE == initGL()) {
        return;
    }

    glutDisplayFunc(display);
    glutReshapeFunc(reshape);

    //����� ����� ��������������������
}

extern void runVisualisation()
{
    // start rendering mainloop
    glutMainLoop();


}

void display()
{
    glutSetWindow(window_id);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();


    //������������ ������� � ������� ������������ ������

    double x = 0;
    double y = 0;
    for (int j = 0; j < dim; j++)
    {
        x = 0;
        for (int i = 0; i < dim; i++)
        {
            if (resource_type[i][j] == 0)
                glColor3ub(230, 230, 250); //lavander (��� �������� �����)
            if (resource_type[i][j] == 1)
                glColor3ub(144, 238, 144); // lightGreen (������������������ ������� �����)
            if (resource_type[i][j] == 2)
                glColor3ub(0, 128, 0); // Green (������������������� ������� �����)

            glRectd(x, y, x + side_size, y + side_size);

            glLineWidth(1);       // ������ �����
            glBegin(GL_LINE_LOOP);
            glColor3d(0, 0, 0);     // ������ ����
            //������� ������
            glVertex2d(x, y);
            glVertex2d(x, y + side_size);
            glVertex2d(x + side_size, y + side_size);
            glVertex2d(x + side_size, y);
            glVertex2d(x, y);
            glEnd();

            x += side_size;
        }
        y += side_size;
    }


    for (int i = 0; i < a_size; i++)
    {
        if (agent_type[i] == 1) //�������� ������
            glColor3ub(255, 0, 0); // ������� ����
        if (agent_type[i] == 2) // ��������
            glColor3ub(0, 0, 255); // ����� ����


        glRectd(x_a[i], y_a[i], x_a[i] + side_size, y_a[i] + side_size);
    }


    //redraw
    glutSwapBuffers();
    glutPostRedisplay();

}


void timer(int = 0)
{
    display();
    glutTimerFunc(1, timer, 0);
}


std::atomic<unsigned int> iterator0 = { 0 };
std::array<std::atomic_int, RUN_COUNT> myArray_id = {}; // ������ ID-�������
FLAMEGPU_INIT_FUNCTION(init_function) {

    std::lock_guard<std::mutex> lock(m);
 
    // ���������� �������� ���������� ������
    flamegpu::HostAgentAPI agent = FLAMEGPU->agent("agent");
    
    std::vector<Cells> high_tech_cells;
    std::vector<Cells> low_tech_cells;

    //������������� �������� ����������� ���������� ������ (��������� �������� ��������� �����)

    double x = 0;
    double y = 0;

    std::vector<Cells> free_cells;

    int cells_count = 0;
    free_cells.clear();
    for (int j = 0; j < dim; j++)
    {
        x = 0;
        for (int i = 0; i < dim; i++)
        {
            if (occupied_cells[i][j] != 1)
                free_cells.push_back(Cells(x, y, i, j));

            x += side_size;
        }
        y += side_size;
    }

      
    //��������� ����� �������-�������
    
    int n = 0;
    if(free_cells.size() > 0)
    { 
        while (n < NumberOfCitizens && free_cells.size() > 0)
        {
            flamegpu::HostNewAgentAPI instance = agent.newAgent();
            int cells_count = free_cells.size();
            int indexx = 0;
            if (cells_count > 0)
                indexx = FLAMEGPU->random.uniform(0, cells_count - 1);
            else
                exit;

            double x1 = free_cells[indexx].x; // ������
            double y1 = free_cells[indexx].y; // �������

            int gender = (int)round(FLAMEGPU->random.uniform<double>());
            int age = (int)FLAMEGPU->random.uniform(1, 80);
            double education_level = (int)log(FLAMEGPU->random.logNormal<double>(2, 1));
            int language_level = 100;
            double comfort_level = round(FLAMEGPU->random.uniform(5, 10));
            int married = 0;
            int kids = 0;

            instance.setVariable<int>("id", getNextID());
            instance.setVariable<float>("x", free_cells[indexx].x);
            instance.setVariable<float>("y", free_cells[indexx].y);
            instance.setVariable<int>("row", free_cells[indexx].r);
            instance.setVariable<int>("clm", free_cells[indexx].c);
            instance.setVariable<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });
            instance.setVariable<int>("type", 1); // ��������
            instance.setVariable<int>("current_state", 1);
            instance.setVariable<int>("previous_state", 1);
            instance.setVariable<int>("gender", gender);
            instance.setVariable<int>("age", age);
            instance.setVariable<double>("education_level", education_level);
            instance.setVariable<int>("language_level", language_level);
            instance.setVariable<double>("comfort_level", comfort_level);
            instance.setVariable<int>("married", married);
            instance.setVariable<int>("kids", kids);
            instance.setVariable<int>("type_resource", 0);
            instance.setVariable<int>("unemployed", 1);
            instance.setVariable<int>("t_arrival", 0);
           // instance.setVariable<int>("CI", -1);
            instance.setVariable<int>("move", 1);


            high_tech_cells.push_back(Cells(free_cells[indexx].x, free_cells[indexx].y, free_cells[indexx].r, free_cells[indexx].c));

            occupied_cells[free_cells[indexx].r][free_cells[indexx].c] = 1;
            if(indexx < free_cells.size())
            free_cells.erase(free_cells.begin() + indexx);
                      
      
            n++;
        }
    }
    //��������� ����� �������-���������
    
    n = 0;
    if(free_cells.size() > 0)
    { 
        while (n < NumberOfMigrants && free_cells.size() > 0)
        {
            flamegpu::HostNewAgentAPI instance = agent.newAgent();
            int cells_count = free_cells.size();
            int indexx = FLAMEGPU->random.uniform(0, cells_count - 1);
            double x1 = free_cells[indexx].x; // ������
            double y1 = free_cells[indexx].y; // �������

            int gender = (int)round(FLAMEGPU->random.uniform<double>());
            int age = (int)FLAMEGPU->random.uniform(1, 50);
            double education_level = 1;
            int language_level = 1;
            double comfort_level = round(FLAMEGPU->random.uniform(0, 5));
            int married = 0;
            int kids = 0;



            instance.setVariable<int>("id", getNextID());
            instance.setVariable<float>("x", free_cells[indexx].x);
            instance.setVariable<float>("y", free_cells[indexx].y);
            instance.setVariable<int>("row", free_cells[indexx].r);
            instance.setVariable<int>("clm", free_cells[indexx].c);
            instance.setVariable<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });
            instance.setVariable<int>("type", 2); // ��������
            instance.setVariable<int>("current_state", 1);
            instance.setVariable<int>("previous_state", 1);
            instance.setVariable<int>("gender", gender);
            instance.setVariable<int>("age", age);
            instance.setVariable<double>("education_level", education_level);
            instance.setVariable<int>("language_level", language_level);
            instance.setVariable<double>("comfort_level", comfort_level);
            instance.setVariable<int>("married", married);
            instance.setVariable<int>("kids", kids);
            instance.setVariable<int>("type_resource", 0);
            instance.setVariable<int>("unemployed", 1);
            instance.setVariable<int>("t_arrival", 0);
            //instance.setVariable<int>("CI", -1);
            instance.setVariable<int>("move", 1);

            low_tech_cells.push_back(Cells(free_cells[indexx].x, free_cells[indexx].y, free_cells[indexx].r, free_cells[indexx].c));

            occupied_cells[free_cells[indexx].r][free_cells[indexx].c] = 1;
            if (indexx < free_cells.size())
            free_cells.erase(free_cells.begin() + indexx);

            n++;
        }
    }
    //��������� ����� ��������
       // ���������� �������� ���������� ������
    flamegpu::HostAgentAPI resource = FLAMEGPU->agent("resources");


    int cnt = 0;
    x = 0;
    y = 0;


    for (int j = 0; j < dim; j++)
    {
        x = 0;
        for (int i = 0; i < dim; i++)
        {
            flamegpu::HostNewAgentAPI instance = resource.newAgent();
            instance.setVariable<int>("id", getNextID()); // 
            instance.setVariable<float>("x", x); // 
            instance.setVariable<float>("y", y); // 
            instance.setVariable<int>("row", i); // 
            instance.setVariable<int>("clm", j); // 
            instance.setVariable<int>("type_resource", 0); // ������ �� ����� �������� �����
            instance.setVariable<int>("time_creation", 0); // ������ �� ����� �������� �����
            instance.setVariable<int>("is_occupied", 1); // �� ��������� ��� ������ ������
            instance.setVariable<unsigned int, 2>("pos", { (unsigned int)i, (unsigned int)j });
          //  instance.setVariable<int>("CI", -1); // ������ �������� �� ���������
            x += side_size;
            cnt++;
        }
        y += side_size;
    }
    //******************************************** ������������� ������� (�����, ���������� ��������)************

        // ������������� �������� ������� *********************************************************************************
    std::vector<Cells>  centroids1;
    if (high_tech_cells.size() > 2)
    {
        // ����� 3-� ��������� ������� � �������� ������� ���������
        int a11 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
        int a21;
        do {
            a21 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
        } while (a21 == a11);
        int a31;
        do {
            a31 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
        } while (a31 == a11 || a31 == a21);


        // ���������� ������� � ������ ������� ���������
        centroids1.push_back(high_tech_cells.at(a11));
        centroids1.push_back(high_tech_cells.at(a21));
        centroids1.push_back(high_tech_cells.at(a31));

        //����� ��������� �������������
        CMeansAlgorithm alg1(high_tech_cells, centroids1, 2);

        //������ ��������� �������-���������
        float c11_x = alg1.getCentersX(0);
        float c11_y = alg1.getCentersY(0);
        float c21_x = alg1.getCentersX(1);
        float c21_y = alg1.getCentersY(1);
        float c31_x = alg1.getCentersX(2);
        float c31_y = alg1.getCentersY(2);

        // ���������� ����� � �������� ����������� ������������ ���������� �������� ���������

        double d1_min = 100000;
        double d2_min = 100000;
        double d3_min = 100000;

        y = 0;
        for (int j = 0; j < dim; j++)
        {
            x = 0;
            for (int i = 0; i < dim; i++)
            {
                double d1 = sqrt(pow(c11_x - x, 2) + pow(c11_y - y, 2));
                double d2 = sqrt(pow(c21_x - x, 2) + pow(c21_y - y, 2));
                double d3 = sqrt(pow(c31_x - x, 2) + pow(c31_y - y, 2));

                if (d1 < d1_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_first_high_tech_cluster", { i, j });
                    d1_min = d1;
                }

                if (d2 < d2_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_second_high_tech_cluster", { i, j });
                    d2_min = d2;
                }

                if (d3 < d3_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_third_high_tech_cluster", { i, j });
                    d3_min = d3;
                }
                x += side_size;
            }
            y += side_size;
        }

        //�������� 
        int r1 = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 0);
        int c1 = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 1);

        int r2 = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 0);
        int c2 = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 1);

        int r3 = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 0);
        int c3 = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 1);

    }

    // ������������� ��������� *********************************************************************************
    std::vector<Cells>  centroids2;
    if (low_tech_cells.size() > 2)
    {
        // ����� 3-� ��������� ������� � �������� ������� ���������
        int a12 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
        int a22;
        do {
            a22 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
        } while (a22 == a12);
        int a32;
        do {
            a32 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
        } while (a32 == a12 || a32 == a22);


        // ���������� ������� � ������ ������� ���������
        centroids2.push_back(low_tech_cells.at(a12));
        centroids2.push_back(low_tech_cells.at(a22));
        centroids2.push_back(low_tech_cells.at(a32));

        //����� ��������� �������������
        CMeansAlgorithm alg2(low_tech_cells, centroids2, 2);

        //������ ��������� �������-���������
        float c12_x = alg2.getCentersX(0);
        float c12_y = alg2.getCentersY(0);
        float c22_x = alg2.getCentersX(1);
        float c22_y = alg2.getCentersY(1);
        float c32_x = alg2.getCentersX(2);
        float c32_y = alg2.getCentersY(2);

        // ���������� ����� � �������� ����������� ������������ ���������� �������� ���������

        double d1_min = 100000;
        double d2_min = 100000;
        double d3_min = 100000;

        y = 0;
        for (int j = 0; j < dim; j++)
        {
            x = 0;
            for (int i = 0; i < dim; i++)
            {
                double d1 = sqrt(pow(c12_x - x, 2) + pow(c12_y - y, 2));
                double d2 = sqrt(pow(c22_x - x, 2) + pow(c22_y - y, 2));
                double d3 = sqrt(pow(c32_x - x, 2) + pow(c32_y - y, 2));

                if (d1 < d1_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_first_low_tech_cluster", { i, j });
                    d1_min = d1;
                }

                if (d2 < d2_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_second_low_tech_cluster", { i, j });
                    d2_min = d2;
                }

                if (d3 < d3_min)
                {
                    FLAMEGPU->environment.setProperty<int, 2>("Center_of_third_low_tech_cluster", { i, j });
                    d3_min = d3;
                }
                x += side_size;
            }
            y += side_size;
        }
    }

    //**********************************************************************************************************************
    int id_thr = std::hash<std::thread::id>{}(std::this_thread::get_id());
    myArray_id[iterator0] = id_thr; // ID ������
    iterator0++; // ������ �������
 }




FLAMEGPU_STEP_FUNCTION(cells_update) {
    std::lock_guard<std::mutex> lock(m);

    auto agent = FLAMEGPU->agent("agent");
    flamegpu::DeviceAgentVector population3 = agent.getPopulationData();

    for (int j = 0; j < dim; j++)
        for (int i = 0; i < dim; i++)
            occupied_cells[i][j] = 0;

    a_size = 0;
    std::vector<Cells> high_tech_cells;
    std::vector<Cells> low_tech_cells;
    high_tech_cells.clear();
    low_tech_cells.clear();

    std::vector<std::vector<int>> agent_in_cell;
    agent_in_cell.clear();
    agent_in_cell.assign(dim, std::vector<int>(dim));
       
    for (int i = 0; i < agent.count(); i++)
    {
        flamegpu::AgentVector::Agent instance = population3[i];

        int r = instance.getVariable<int>("row");
        int c = instance.getVariable<int>("clm");
        occupied_cells[r][c] = 1;

        x_a[i] = instance.getVariable<float>("x");
        y_a[i] = instance.getVariable<float>("y");
        agent_type[i] = instance.getVariable<int>("type");

        if (agent_type[i] == 1)
        {
            high_tech_cells.push_back(Cells(x_a[i], y_a[i], r, c));
            agent_in_cell[r][c] = 1;
        }
            
        if (agent_type[i] == 2)
        {
            low_tech_cells.push_back(Cells(x_a[i], y_a[i], r, c));
            agent_in_cell[r][c] = 2;
        }

        a_size++;
    }


    flamegpu::HostAgentAPI resource = FLAMEGPU->agent("resources");
    flamegpu::DeviceAgentVector population2 = resource.getPopulationData();

    for (int i = 0; i < resource.count(); i++)
    {
        flamegpu::AgentVector::Agent instance = population2[i];
        if (instance.getVariable<int>("id") != 0)
        {
            float x = instance.getVariable<float>("x");
            float y = instance.getVariable<float>("y");

            int r = instance.getVariable<int>("row");
            int c = instance.getVariable<int>("clm");
            resource_type[r][c] = instance.getVariable<int>("type_resource");
            int is_occupied = instance.getVariable<int>("is_occupied");
        }
    }
    
    //population2.syncChanges();
    population2.purgeCache();
    population3.purgeCache();

    //���������� �������������
    uint8_t Frequency = FLAMEGPU->environment.getProperty<uint8_t>("Frequency_work_places_creation");

    int year = FLAMEGPU->getStepCounter() + 1; // ���, �������������� �������� ������� ����
    if ( year % Frequency == 0 ) // 
    {
        //******************************************** ������������� ������� (�����, ���������� ��������)************

    // ������������� �������� ������� *********************************************************************************
        std::vector<Cells>  centroids1;
        if (high_tech_cells.size() > 2)
        {
            // ����� 3-� ��������� ������� � �������� ������� ���������
            int a11 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
            int a21;
            do {
                a21 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
            } while (a21 == a11);
            int a31;
            do {
                a31 = (int)(FLAMEGPU->random.uniform<double>() * high_tech_cells.size());
            } while (a31 == a11 || a31 == a21);


            // ���������� ������� � ������ ������� ���������
            centroids1.push_back(high_tech_cells.at(a11));
            centroids1.push_back(high_tech_cells.at(a21));
            centroids1.push_back(high_tech_cells.at(a31));

            //����� ��������� �������������
            CMeansAlgorithm alg1(high_tech_cells, centroids1, 2);

            //������ ��������� �������-���������
            float c11_x = alg1.getCentersX(0);
            float c11_y = alg1.getCentersY(0);
            float c21_x = alg1.getCentersX(1);
            float c21_y = alg1.getCentersY(1);
            float c31_x = alg1.getCentersX(2);
            float c31_y = alg1.getCentersY(2);

            // ���������� ����� � �������� ����������� ������������ ���������� �������� ���������

            double d1_min = 100000;
            double d2_min = 100000;
            double d3_min = 100000;

            double y = 0;
            for (int j = 0; j < dim; j++)
            {
                double x = 0;
                for (int i = 0; i < dim; i++)
                {
                    double d1 = sqrt(pow(c11_x - x, 2) + pow(c11_y - y, 2));
                    double d2 = sqrt(pow(c21_x - x, 2) + pow(c21_y - y, 2));
                    double d3 = sqrt(pow(c31_x - x, 2) + pow(c31_y - y, 2));

                    if (d1 < d1_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_first_high_tech_cluster", { i, j });
                        d1_min = d1;
                    }

                    if (d2 < d2_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_second_high_tech_cluster", { i, j });
                        d2_min = d2;
                    }

                    if (d3 < d3_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_third_high_tech_cluster", { i, j });
                        d3_min = d3;
                    }
                    x += side_size;
                }
                y += side_size;
            }

            //�������� 
            int r1 = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 0);
            int c1 = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 1);

            int r2 = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 0);
            int c2 = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 1);

            int r3 = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 0);
            int c3 = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 1);

        }

        // ������������� ��������� *********************************************************************************
        std::vector<Cells>  centroids2;
        if (low_tech_cells.size() > 2)
        {
            // ����� 3-� ��������� ������� � �������� ������� ���������
            int a12 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
            int a22;
            do {
                a22 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
            } while (a22 == a12);
            int a32;
            do {
                a32 = (int)(FLAMEGPU->random.uniform<double>() * low_tech_cells.size());
            } while (a32 == a12 || a32 == a22);


            // ���������� ������� � ������ ������� ���������
            centroids2.push_back(low_tech_cells.at(a12));
            centroids2.push_back(low_tech_cells.at(a22));
            centroids2.push_back(low_tech_cells.at(a32));

            //����� ��������� �������������
            CMeansAlgorithm alg2(low_tech_cells, centroids2, 2);

            //������ ��������� �������-���������
            float c12_x = alg2.getCentersX(0);
            float c12_y = alg2.getCentersY(0);
            float c22_x = alg2.getCentersX(1);
            float c22_y = alg2.getCentersY(1);
            float c32_x = alg2.getCentersX(2);
            float c32_y = alg2.getCentersY(2);

            // ���������� ����� � �������� ����������� ������������ ���������� �������� ���������

            double d1_min = 100000;
            double d2_min = 100000;
            double d3_min = 100000;

            double y = 0;
            for (int j = 0; j < dim; j++)
            {
               double x = 0;
                for (int i = 0; i < dim; i++)
                {
                    double d1 = sqrt(pow(c12_x - x, 2) + pow(c12_y - y, 2));
                    double d2 = sqrt(pow(c22_x - x, 2) + pow(c22_y - y, 2));
                    double d3 = sqrt(pow(c32_x - x, 2) + pow(c32_y - y, 2));

                    if (d1 < d1_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_first_low_tech_cluster", { i, j });
                        d1_min = d1;
                    }

                    if (d2 < d2_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_second_low_tech_cluster", { i, j });
                        d2_min = d2;
                    }

                    if (d3 < d3_min)
                    {
                        FLAMEGPU->environment.setProperty<int, 2>("Center_of_third_low_tech_cluster", { i, j });
                        d3_min = d3;
                    }
                    x += side_size;
                }
                y += side_size;
            }
        }
        //*******************************************************************************************************
    }

    double DSI = Dunkan_segregation_index(high_tech_cells.size(), low_tech_cells.size(), agent_in_cell);

    FLAMEGPU->environment.setProperty<float>("Total_DSI", FLAMEGPU->environment.getProperty<float>("Total_DSI") + DSI);

    double DSI_avg = FLAMEGPU->environment.getProperty<float>("Total_DSI") / (FLAMEGPU->getStepCounter() + 1);
    FLAMEGPU->environment.setProperty<float>("Average_DSI", DSI_avg); // ������� �������� ������� DSI
}

std::array<std::atomic_int, RUN_COUNT> myArray_test1 = {};
std::array<std::atomic_int, RUN_COUNT> myArray_test2 = {};
std::array<std::atomic_int, RUN_COUNT> myArray_test3 = {};
FLAMEGPU_STEP_FUNCTION(BasicOutput) {

   std::lock_guard<std::mutex> lock(m);

    /* �������� ���� ��� �������� ������� */

    flamegpu::HostAgentAPI resource = FLAMEGPU->agent("resources");
    int id_thr = std::hash<std::thread::id>{}(std::this_thread::get_id());
    for(int i = 0 ; i < RUN_COUNT; i++)
    { 
       if(myArray_id[i]== id_thr)
                { 
                myArray_test1[i] = FLAMEGPU->getStepCounter(); // ��������� ������� �������
                myArray_test2[i] = FLAMEGPU->agent("agent").count(); // ���������� �������
                myArray_test3[i] = resource.count<int>("is_occupied", 0);// ���������� ��������� �����
                }
    }
   
   
    if (FLAMEGPU->getStepCounter() > 0)
    {
        flamegpu::HostAgentAPI resource = FLAMEGPU->agent("resources");
        // ���������� �������� ���������� ������
        auto agent = FLAMEGPU->agent("agent");

        int count_free_cells = resource.count<int>("is_occupied", 0); // ���������� ��������� �����
        int count_occupied_cells = resource.count<int>("is_occupied", 1); // ���������� ����������� �����
        int agents_count = agent.count(); // ���������� �������


        //auto location = FLAMEGPU->environment.getMacroProperty<uint32_t, 100, 100>("occupied_cells");

        double x = 0;
        double y = 0;

        std::vector<Cells> free_cells;

        int cells_count = 0;
        free_cells.clear();
        for (int j = 0; j < dim; j++)
        {
            x = 0;
            for (int i = 0; i < dim; i++)
            {
                if (occupied_cells[i][j] != 1) // ������ ������ ��������
                    free_cells.push_back(Cells(x, y, i, j));

                x += side_size;
            }
            y += side_size;
        }

        int mig = agent.count<int>("type", 2);

        // Create NEW_AGENT_COUNT new 'agent' agents with 'x' set to 1.0f
        float share_of_new_migrants = FLAMEGPU->environment.getProperty<double>("Share_of_new_migrants");

        //�������� ����� �������-���������
        

        int agent_migrants_count = agent.count<int>("type", 2);
        int agent_native_count = agent.count<int>("type", 1);
        int agent_zero_count = agent.count<int>("type", 0);
        int new_migrants = (int)round(agent_migrants_count * share_of_new_migrants);
        int n = 0;
      
        if(free_cells.size() > 0)
        { 
            while (n < new_migrants && free_cells.size() > 0) // 
            {
                //int check_agents =  agent.count<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });

                flamegpu::HostNewAgentAPI instance = agent.newAgent();

                int cells_count = free_cells.size();
                int indexx = FLAMEGPU->random.uniform(0, cells_count - 1);
                double x1 = free_cells[indexx].x; // ������
                double y1 = free_cells[indexx].y; // �������

                int gender = (int)round(FLAMEGPU->random.uniform<double>());
                int age = (int)FLAMEGPU->random.uniform(1, 50);
                double education_level = 1;
                int language_level = 1;
                double comfort_level = round(FLAMEGPU->random.uniform(0, 5));
                int married = 0;
                int kids = 0;


                instance.setVariable<int>("id", getNextID());
                instance.setVariable<float>("x", free_cells[indexx].x);
                instance.setVariable<float>("y", free_cells[indexx].y);
                instance.setVariable<int>("row", free_cells[indexx].r);
                instance.setVariable<int>("clm", free_cells[indexx].c);
                instance.setVariable<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });
                instance.setVariable<int>("type", 2); // ��������
                instance.setVariable<int>("current_state", 1);
                instance.setVariable<int>("previous_state", 1);
                instance.setVariable<int>("gender", gender);
                instance.setVariable<int>("age", age);
                instance.setVariable<double>("education_level", education_level);
                instance.setVariable<int>("language_level", language_level);
                instance.setVariable<double>("comfort_level", comfort_level);
                instance.setVariable<int>("married", married);
                instance.setVariable<int>("kids", kids);
                instance.setVariable<int>("type_resource", 0);
                instance.setVariable<int>("unemployed", 1);
                instance.setVariable<int>("t_arrival", FLAMEGPU->getStepCounter());
                //instance.setVariable<int>("CI", -1);
                instance.setVariable<int>("move", 1);

                occupied_cells[free_cells[indexx].r][free_cells[indexx].c] = 1;
                if (indexx < free_cells.size())
                free_cells.erase(free_cells.begin() + indexx);
            
                n++;
            }
        }
        //�������� ����� �������-�������
        agent_migrants_count = agent.count<int>("type", 2);
        agent_native_count = agent.count<int>("type", 1);
        agent_zero_count = agent.count<int>("type", 0);


        int number_of_natives_should_be_born = agent.count<int>("kids_should_be_born_native", 1);
               
        n = 0;
        if(free_cells.size() > 0)
        {
            while (n < number_of_natives_should_be_born && free_cells.size() > 0) // 
            {
                flamegpu::HostNewAgentAPI instance = agent.newAgent();

                int cells_count = free_cells.size();
                int indexx = 0;
                if (cells_count > 0)
                    indexx = FLAMEGPU->random.uniform(0, cells_count - 1);
                else
                    exit;

                double x1 = free_cells[indexx].x; // ������
                double y1 = free_cells[indexx].y; // �������

                int gender = (int)round(FLAMEGPU->random.uniform<double>());
                int age = 0;
                double education_level = 1;
                int language_level = 100;
                double comfort_level = 10;
                int married = 0;
                int kids = 0;


                instance.setVariable<int>("id", getNextID());
                instance.setVariable<float>("x", free_cells[indexx].x);
                instance.setVariable<float>("y", free_cells[indexx].y);
                instance.setVariable<int>("row", free_cells[indexx].r);
                instance.setVariable<int>("clm", free_cells[indexx].c);
                instance.setVariable<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });
                instance.setVariable<int>("type", 1); // ��������
                instance.setVariable<int>("current_state", 1);
                instance.setVariable<int>("previous_state", 1);
                instance.setVariable<int>("gender", gender);
                instance.setVariable<int>("age", age);
                instance.setVariable<double>("education_level", education_level);
                instance.setVariable<int>("language_level", language_level);
                instance.setVariable<double>("comfort_level", comfort_level);
                instance.setVariable<int>("married", married);
                instance.setVariable<int>("kids", kids);
                instance.setVariable<int>("type_resource", 0);
                instance.setVariable<int>("unemployed", 1);
                instance.setVariable<int>("t_arrival", FLAMEGPU->getStepCounter());
              //  instance.setVariable<int>("CI", -1);
                instance.setVariable<int>("move", 1);

                occupied_cells[free_cells[indexx].r][free_cells[indexx].c] = 1;
                if (indexx < free_cells.size())
                free_cells.erase(free_cells.begin() + indexx);

                n++;
            }
        }

        //�������� ����� �������-���������
        agent_migrants_count = agent.count<int>("type", 2);
        agent_native_count = agent.count<int>("type", 1);
        agent_zero_count = agent.count<int>("type", 0);

        int number_of_migrants_should_be_born = agent.count<int>("kids_should_be_born_migrant", 1);

        n = 0;
        if(free_cells.size() > 0)
        { 
            while (n < number_of_migrants_should_be_born && free_cells.size() > 0) // 
            {
                flamegpu::HostNewAgentAPI instance = agent.newAgent();

                int cells_count = free_cells.size();
                int indexx = 0;
                if(cells_count > 0)
                    indexx = FLAMEGPU->random.uniform(0, cells_count - 1);
                else
                    exit;

                double x1 = free_cells[indexx].x; // ������
                double y1 = free_cells[indexx].y; // �������

                int gender = (int)round(FLAMEGPU->random.uniform<double>());
                int age = 0;
                double education_level = 0;
                int language_level = 0;
                double comfort_level = 10;
                int married = 0;
                int kids = 0;


                instance.setVariable<int>("id", getNextID());
                instance.setVariable<float>("x", free_cells[indexx].x);
                instance.setVariable<float>("y", free_cells[indexx].y);
                instance.setVariable<int>("row", free_cells[indexx].r);
                instance.setVariable<int>("clm", free_cells[indexx].c);
                instance.setVariable<unsigned int, 2>("pos", { (unsigned int)free_cells[indexx].r, (unsigned int)free_cells[indexx].c });
                instance.setVariable<int>("type", 2); // ��������
                instance.setVariable<int>("current_state", 1);
                instance.setVariable<int>("previous_state", 1);
                instance.setVariable<int>("gender", gender);
                instance.setVariable<int>("age", age);
                instance.setVariable<double>("education_level", education_level);
                instance.setVariable<int>("language_level", language_level);
                instance.setVariable<double>("comfort_level", comfort_level);
                instance.setVariable<int>("married", married);
                instance.setVariable<int>("kids", kids);
                instance.setVariable<int>("type_resource", 0);
                instance.setVariable<int>("unemployed", 1);
                instance.setVariable<int>("t_arrival", FLAMEGPU->getStepCounter());
             //   instance.setVariable<int>("CI", -1);
                instance.setVariable<int>("move", 1);

                occupied_cells[free_cells[indexx].r][free_cells[indexx].c] = 1;
                if(indexx < free_cells.size())
                free_cells.erase(free_cells.begin() + indexx);

                n++;
            }
        }
        // ������ ��������� ������������������ �����������

        //���� � ��������� ���� ��� �������������������� �������, �������� �������
        double p1 = 1000;  //������� ���� �� ��������� ����������������� �������� ���������

        double A_r1 = 1.01;
        double a_r1 = 0.1;
        double b_r1 = 0.9;


        double G_r1 = 0.1;
        double c_r1 = 0.1;
        double d_r1 = 0.9;


        //���� � ��������� ���� ��� ������������ �������, �������� �������
        double p2 = 300; //������� ���� �� ��������� ������������������ �������� ���������

        double A_r2 = 1;
        double a_r2 = 0.2;
        double b_r2 = 0.8;

        double G_r2 = 1.5;
        double c_r2 = 0.5;
        double d_r2 = 0.5;

        int agents_in_high_sectors = agent.count<int>("type_resource", 2);
        int agents_in_low_sectors = agent.count<int>("type_resource", 1);
        int unemployment_agents = agent.count<int>("unemployed", 1);
        int agent_pensioners = agent.count<int>("is_pensioner", 1);
        int number_of_assimilated_agents = agent.count<int>("is_assimilated", 1);

        float Expenditure_on_pensioners = 2400; // �������� ������ ������ � ��� �� 1 ����������
        float Expenditure_on_unemploymenters = 2000; // �������� ������ ������ � ��� �� 1 ������������

        // 1.2 � 0.8 ��� ������� ���������������� ������ � ������������������� � ������������������ �������� ���������
        float V = p1 * (1.2 * agents_in_high_sectors) + p2 * (0.8 * agents_in_low_sectors) -
            agent_pensioners * Expenditure_on_pensioners -
            unemployment_agents * Expenditure_on_unemploymenters;

        float GE =
            (agent_pensioners * Expenditure_on_pensioners +
                unemployment_agents * Expenditure_on_unemploymenters +
                agents_in_low_sectors * 12000) / 1000000; // 12000 - ��������� �������� 1 �������� ����� ��� ���������




        float Rate_V = 0;
        float Rate_GE = 0;

        if (FLAMEGPU->getStepCounter() == 1)
        {
            Rate_V = 1.05;
            Rate_GE = 1.01;
        }

        else
        {
            Rate_V = V / FLAMEGPU->environment.getProperty<float>("GDP"); // ���
            Rate_GE = GE / FLAMEGPU->environment.getProperty<float>("GE"); //���. �������
        }

        Rate_V += FLAMEGPU->environment.getProperty<float>("GDP_rate_total");
        Rate_GE += FLAMEGPU->environment.getProperty<float>("GE_rate_total");
        FLAMEGPU->environment.setProperty<float>("GDP_rate_total", Rate_V);
        FLAMEGPU->environment.setProperty<float>("GE_rate_total", Rate_GE);

        float average_Rate_GDP = FLAMEGPU->environment.getProperty<float>("GDP_rate_total") / FLAMEGPU->getStepCounter(); // ������� ���� ����� ���
        float average_Rate_GE = FLAMEGPU->environment.getProperty<float>("GE_rate_total") / FLAMEGPU->getStepCounter(); // ������� ���� ����� ��������������� ��������

        FLAMEGPU->environment.setProperty<float>("Average_GDP_rate", average_Rate_GDP);
        FLAMEGPU->environment.setProperty<float>("Average_Government_Expenditure_rate", average_Rate_GE);

        FLAMEGPU->environment.setProperty<float>("GDP", V);
        FLAMEGPU->environment.setProperty<float>("GE", GE);

    }


    if (out.is_open() && VIS_MODE == 1) // �������� ����������� ��� ���������� �������
    {
        if (FLAMEGPU->getStepCounter() == 0)
        {
            out << "Share_of_new_migrants" <<
                ";" << "Expenditure_on_education_share" <<
                ";" << "Life_age_high_technology_work_places" <<
                ";" << "Life_age_low_technology_work_places" <<
                ";" << "Frequency_work_places_creation" <<
                ";" << "Average_life_time_of_natives" <<
                ";" << "Average_life_time_of_migrants" <<
                ";" << "Age_for_married_and_kids_birth_of_natives" <<
                ";" << "Age_for_married_and_kids_birth_of_migrants" <<
                ";" << "Minimum_comfort_level_of_natives" <<
                ";" << "Minimum_comfort_level_of_migrants" <<
                ";" << "Pension_age" <<
                ";" << "Method_work_places_creation" <<

                ";" << "Total_count_of_agents" <<
                ";" << "Share_of_non-assimilated_migrants" <<
                ";" << "Number_of_assimilated_migrants" <<
                ";" << "Averaged_time_for_assimilation" <<
                ";" << "Duncan_Segregation_Index" <<
                ";" << "Average_GDP_rate" <<
                ";" << "Average_Government_Expenditure_rate" << std::endl;
        }

        //�������� ���������� ����������� � ��������
        if (FLAMEGPU->getStepCounter() > 0)
        {
            auto agent = FLAMEGPU->agent("agent");
            
            int agent_count = 0;
            if (agent.count() > 0)
                agent_count = agent.count();

            float t_assim = 0;
            if (agent.count<int>("type", 2) > 0)
                t_assim = (double)(agent.sum<double>("Time_for_assimilation") / agent.count<int>("type", 2));

            int number_of_nonassimilated = agent.count<int>("type", 2);
            float share_of_nonassimilated = (float)number_of_nonassimilated / agent_count;


            out << FLAMEGPU->environment.getProperty<double>("Share_of_new_migrants") <<
                ";" << FLAMEGPU->environment.getProperty<double>("Expenditure_on_education_share") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Life_age_high_technology_work_places") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Life_age_low_technology_work_places") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Frequency_work_places_creation") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_natives") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_migrants") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_natives") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_migrants") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_natives") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_migrants") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Pension_age") <<
                ";" << (int)FLAMEGPU->environment.getProperty<uint8_t>("Method_work_places_creation") <<

                ";" << agent.count() <<
                ";" << share_of_nonassimilated <<
                ";" << agent.count<int>("is_assimilated", 1) <<
                ";" << t_assim <<
                ";" << FLAMEGPU->environment.getProperty<float>("Average_DSI") <<
                ";" << FLAMEGPU->environment.getProperty<float>("Average_GDP_rate") <<
                ";" << FLAMEGPU->environment.getProperty<float>("Average_Government_Expenditure_rate") << std::endl;
        }

        if (FLAMEGPU->getStepCounter() == TIME_STOP - 1)
            out.close();

    }

}


std::array<std::atomic_int, RUN_COUNT> myArray1 = {};
std::array<std::atomic_int, RUN_COUNT> myArray2 = {};
std::array<std::atomic_int, RUN_COUNT> myArray3 = {};
std::array<std::atomic_int, RUN_COUNT> myArray4 = {};
std::array<std::atomic_int, RUN_COUNT> myArray5 = {};
std::array<std::atomic_int, RUN_COUNT> myArray6 = {};
std::array<std::atomic_int, RUN_COUNT> myArray7 = {};
std::array<std::atomic_int, RUN_COUNT> myArray8 = {};
std::array<std::atomic_int, RUN_COUNT> myArray9 = {};
std::array<std::atomic_int, RUN_COUNT> myArray10 = {};
std::array<std::atomic_int, RUN_COUNT> myArray11 = {};
std::array<std::atomic_int, RUN_COUNT> myArray12 = {};
std::array<std::atomic_int, RUN_COUNT> myArray13 = {};
std::array<std::atomic_int, RUN_COUNT> myArray14 = {};
std::array<std::atomic_int, RUN_COUNT> myArray15 = {};
std::array<std::atomic_int, RUN_COUNT> myArray16 = {};
std::array<std::atomic_int, RUN_COUNT> myArray17 = {};
std::array<std::atomic_int, RUN_COUNT> myArray18 = {};
std::array<std::atomic_int, RUN_COUNT> myArray19 = {};
std::array<std::atomic_int, RUN_COUNT> myArray20 = {};

std::atomic<unsigned int> iterator = { 0 };
FLAMEGPU_EXIT_CONDITION(exit_condition) {
    std::lock_guard<std::mutex> lock(m);

	if (FLAMEGPU->getStepCounter() >= TIME_STOP - 1)
	{
		
		auto agent = FLAMEGPU->agent("agent");

		flamegpu::DeviceAgentVector population(agent.getPopulationData());
		a_size = 0;
		int counter = 0;
		double assim_t = 0.0;

		for (int i = 0; i < agent.count(); i++)
		{
			flamegpu::AgentVector::Agent instance = population[i];
			x_a[i] = instance.getVariable<float>("x");
			y_a[i] = instance.getVariable<float>("y");
			agent_type[i] = instance.getVariable<int>("type");

			if (instance.getVariable<double>("Time_for_assimilation") > 0)
			{
				assim_t += instance.getVariable<double>("Time_for_assimilation");
				counter++;
			}

			a_size++;
		}



		int agent_migrants_count = agent.count<int>("type", 2); // ���������� ���������
		float migrants_share = (float)agent_migrants_count / agent.count(); // ���� ���������

		float average_assim_time = 0;
		if (agent_migrants_count > 0)
			average_assim_time = (double)agent.sum<double>("Time_for_assimilation") / agent_migrants_count; // ������� ���� �� �����������

		if (counter > 0)
			assim_t = assim_t / counter; //  ������� ����� �� ����������� (���������� ������)

		population.syncChanges();
		population.purgeCache();
		
		
		myArray1[iterator] = (int)(FLAMEGPU->environment.getProperty<double>("Share_of_new_migrants") * 100000);
		myArray2[iterator] = (int)(FLAMEGPU->environment.getProperty<double>("Expenditure_on_education_share") * 100000);
		myArray3[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Life_age_high_technology_work_places");
		myArray4[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Life_age_low_technology_work_places");
		myArray5[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Frequency_work_places_creation");
		myArray6[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_natives");
		myArray7[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_migrants");
        myArray8[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_natives");
		myArray9[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_migrants");
		myArray10[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_natives");
		myArray11[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_migrants");
		myArray12[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Pension_age");
        myArray13[iterator] = FLAMEGPU->environment.getProperty<uint8_t>("Method_work_places_creation");
        
        myArray14[iterator] = agent.count();
        myArray15[iterator] = (int)(migrants_share*100000);
        myArray16[iterator] = agent.count<int>("is_assimilated", 1);;
        myArray17[iterator] = (int)(assim_t * 100000);
        myArray18[iterator] = (int)(FLAMEGPU->environment.getProperty<float>("Average_DSI") * 100000);
        myArray19[iterator] = (int)(FLAMEGPU->environment.getProperty<float>("Average_GDP_rate") * 100000);
        myArray20[iterator] = (int)(FLAMEGPU->environment.getProperty<float>("Average_Government_Expenditure_rate") * 100000);

		iterator++;
		return  flamegpu::EXIT;  // End the simulation here
	}
    else
        return  flamegpu::CONTINUE;  // Continue the simulation
}


FLAMEGPU_AGENT_FUNCTION(all_agents, flamegpu::MessageNone, flamegpu::MessageArray2D)
{
    //�������� ������ � ������ ��������� ������
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<int>("row", FLAMEGPU->getVariable<int>("row"));
    FLAMEGPU->message_out.setVariable<int>("clm", FLAMEGPU->getVariable<int>("clm"));
    FLAMEGPU->message_out.setVariable<int>("move", FLAMEGPU->getVariable<int>("move"));
    FLAMEGPU->message_out.setVariable<int>("current_state", FLAMEGPU->getVariable<int>("current_state"));
    FLAMEGPU->message_out.setVariable<int>("type", FLAMEGPU->getVariable<int>("type"));
    FLAMEGPU->message_out.setVariable<int>("married", FLAMEGPU->getVariable<int>("married"));
    FLAMEGPU->message_out.setVariable<int>("gender", FLAMEGPU->getVariable<int>("gender"));
    FLAMEGPU->message_out.setVariable<double>("education_level", FLAMEGPU->getVariable<double>("education_level"));

    FLAMEGPU->message_out.setIndex(FLAMEGPU->getVariable<unsigned int, 2>("pos", 0), FLAMEGPU->getVariable<unsigned int, 2>("pos", 1));

    return  flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(all_resources, flamegpu::MessageNone, flamegpu::MessageArray2D)
{
    //�������� ������ � ������ �������
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<int>("type_resource", FLAMEGPU->getVariable<int>("type_resource"));
    FLAMEGPU->message_out.setVariable<int>("is_occupied", FLAMEGPU->getVariable<int>("is_occupied"));
    FLAMEGPU->message_out.setVariable<int>("row", FLAMEGPU->getVariable<int>("row"));
    FLAMEGPU->message_out.setVariable<int>("clm", FLAMEGPU->getVariable<int>("clm"));
    FLAMEGPU->message_out.setVariable<int>("request_agent", FLAMEGPU->getVariable<int>("request_agent"));
    FLAMEGPU->message_out.setIndex(FLAMEGPU->getVariable<unsigned int, 2>("pos", 0), FLAMEGPU->getVariable<unsigned int, 2>("pos", 1));
    
    return  flamegpu::ALIVE;
}


//�������� ������� "�������" �������
FLAMEGPU_AGENT_FUNCTION(check_all_agents, flamegpu::MessageArray2D, flamegpu::MessageNone)
{
    int flag = 0;

    const unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    const unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    //������ 1x1
    // ������ ��������� ������ �� ������, �������������� � ��� �� �����������
    const auto message = FLAMEGPU->message_in.at(my_x, my_y); 
    
        if (message.getVariable<int>("row") == FLAMEGPU->getVariable<int>("row") &&
            message.getVariable<int>("clm") == FLAMEGPU->getVariable<int>("clm") &&
            message.getVariable<int>("id") != FLAMEGPU->getVariable<int>("id"))
        {
            flag = 1; // ��������� "�������" �����
        }
    
    if (flag == 0)
        return  flamegpu::ALIVE;
    else
    {
        printf("Agent with ID is %i is lost\n", FLAMEGPU->getVariable<int>("id"));
        return  flamegpu::DEAD;
    }
}


FLAMEGPU_AGENT_FUNCTION(workplaces_creation, flamegpu::MessageArray2D, flamegpu::MessageNone)
{

    uint8_t method = FLAMEGPU->environment.getProperty<uint8_t>("Method_work_places_creation");
    uint8_t Frequency = FLAMEGPU->environment.getProperty<uint8_t>("Frequency_work_places_creation");
    unsigned int my_x = 0;
    unsigned int my_y = 0;



    if (method == 1 && FLAMEGPU->getStepCounter() % Frequency == 0) // ����������� � ��������� �������� ������� ���� c �������� ��������������
    {
        double p = FLAMEGPU->random.uniform<double>();

        //�������� ��������� ������� ���� � ��������� ������������
        if ((FLAMEGPU->getVariable<int>("is_occupied") == 0 || FLAMEGPU->getStepCounter() == 0) &&
            FLAMEGPU->getVariable<int>("type_resource") == 0 && p > 0.7)
        {
            FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        else if ((FLAMEGPU->getVariable<int>("is_occupied") == 0 || FLAMEGPU->getStepCounter() == 0) &&
            FLAMEGPU->getVariable<int>("type_resource") == 0 &&
            p > 0.6 && p <= 0.7)
        {
            FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // 
        }
    }

    if (method == 2 && FLAMEGPU->getStepCounter() % Frequency == 0) // ���������-��������������� �������� ����� ������� ����
    {
       // ������������������� ������� ����� ***************************************************************************

        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_first_high_tech_cluster", 1);
        int S1 = FLAMEGPU->environment.getProperty<int>("Size_of_high_tech_cluster"); // ����������� ���������-������� ����
        int S2 = FLAMEGPU->environment.getProperty<int>("Size_of_low_tech_cluster");

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }
       
        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1)) 
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }


        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_second_high_tech_cluster", 1);

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1))
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }


        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_third_high_tech_cluster", 1);

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1))
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 2); // ����� ������������������� ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }


        // ������������������ ������� ����� **********************************************************************
        
        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_first_low_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_first_low_tech_cluster", 1);

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1))
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }

        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_second_low_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_second_low_tech_cluster", 1);

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1))
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }

        my_x = FLAMEGPU->environment.getProperty<int>("Center_of_third_low_tech_cluster", 0);  // ���������� ������ ��������
        my_y = FLAMEGPU->environment.getProperty<int>("Center_of_third_low_tech_cluster", 1);

        if (FLAMEGPU->getVariable<int>("row") == my_x && FLAMEGPU->getVariable<int>("clm") == my_y)
        {
            FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
            FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
        }

        for (const auto& message : FLAMEGPU->message_in.wrap(FLAMEGPU->getVariable<int>("row"), FLAMEGPU->getVariable<int>("clm"), S1))
        {
            if (message.getVariable<int>("row") == my_x && message.getVariable<int>("clm") == my_y)
            {
                FLAMEGPU->setVariable<int>("type_resource", 1); // ����� ������������������ ������
                FLAMEGPU->setVariable<int>("time_creation", FLAMEGPU->getStepCounter()); // ����� �������� �������
            }
        }

    }

    // ������� ������� ���� ��� ������� ���������� ���������� ������ ��������
    uint8_t age_max_high = FLAMEGPU->environment.getProperty<uint8_t>("Life_age_high_technology_work_places");
    uint8_t age_max_low = FLAMEGPU->environment.getProperty<uint8_t>("Life_age_low_technology_work_places");

    if (FLAMEGPU->getStepCounter() - FLAMEGPU->getVariable<int>("time_creation") >= age_max_high && FLAMEGPU->getVariable<int>("type_resource") == 2)
    {
        FLAMEGPU->setVariable<int>("type_resource", 0);
        FLAMEGPU->setVariable<int>("time_creation", 0);
    }

    if (FLAMEGPU->getStepCounter() - FLAMEGPU->getVariable<int>("time_creation") >= age_max_low && FLAMEGPU->getVariable<int>("type_resource") == 1)
    {
        FLAMEGPU->setVariable<int>("type_resource", 0);
        FLAMEGPU->setVariable<int>("time_creation", 0);
    }
    return  flamegpu::ALIVE;
}


//������� ���������� ��������� ����� (�������)
FLAMEGPU_AGENT_FUNCTION(update_cell, flamegpu::MessageArray2D, flamegpu::MessageNone) {

    int flag = 0;

    const unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    const unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    FLAMEGPU->setVariable<int>("is_occupied", 0); // �� ��������� ��� ������ ��������

    const auto message = FLAMEGPU->message_in.at(my_x, my_y); 
        
    if ((message.getVariable<int>("row") == FLAMEGPU->getVariable<int>("row") &&
                message.getVariable<int>("clm") == FLAMEGPU->getVariable<int>("clm")))
            {
                flag = 1;
            }
       
   // auto swap = FLAMEGPU->environment.getMacroProperty<uint32_t, 100, 100>("occupied_cells");
   
    if (flag == 0 && FLAMEGPU->getVariable<int>("waiting_occupation") != 1)
    {
        FLAMEGPU->setVariable<int>("is_occupied", 0);
    //    swap[my_x][my_y].exchange(0);
    }
    if (flag == 1)
    {
        FLAMEGPU->setVariable<int>("is_occupied", 1);
    //    swap[my_x][my_y].exchange(1);
    }


    return  flamegpu::ALIVE;
}

//������� �������� ����, ��� ����� �������� ������ ������
FLAMEGPU_AGENT_FUNCTION(check_cell, flamegpu::MessageArray2D, flamegpu::MessageNone) {

    unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);


    const auto message = FLAMEGPU->message_in.at(my_x, my_y);

        if (message.getVariable<int>("row") == FLAMEGPU->getVariable<int>("row") &&
            message.getVariable<int>("clm") == FLAMEGPU->getVariable<int>("clm"))
        {
            //������ ����� �������� ��� ������
            FLAMEGPU->setVariable<int>("type_resource", message.getVariable<int>("type_resource"));


            if (FLAMEGPU->getVariable<int>("type_resource") == 0)
                FLAMEGPU->setVariable<int>("unemployed", 1); // ����� �� ����� ������


            if (FLAMEGPU->getVariable<int>("type_resource") != 0)
                FLAMEGPU->setVariable<int>("unemployed", 0); // ����� ����� ������

        }

    return  flamegpu::ALIVE;
}



//������� ������ ���������� �������������� ������� ������-�������
FLAMEGPU_AGENT_FUNCTION(agent_to_agent_contacts, flamegpu::MessageArray2D, flamegpu::MessageNone) {


    unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    int my_type = FLAMEGPU->getVariable<int>("type");


    //������-���������
    for (const auto& message : FLAMEGPU->message_in(my_x, my_y))
    {

        int r1 = message.getVariable<int>("row");
        int c1 = message.getVariable<int>("clm");
        int r2 = FLAMEGPU->getVariable<int>("row");
        int c2 = FLAMEGPU->getVariable<int>("clm");

        int type_m = message.getVariable<int>("type");

        // ������-���������
        if (((r1 == r2 - 1 || r1 == r2 + 1) && c1 == c2 ||
            (r1 == r2 - 1 || r1 == r2 + 1) && c1 == c2 - 1 ||
            (r1 == r2 - 1 || r1 == r2 + 1) && c1 == c2 + 1) ||

            ((c1 == c2 - 1 || c1 == c2 + 1) && r1 == r2 ||
                (c1 == c2 - 1 || c1 == c2 + 1) && r1 == r2 - 1 ||
                (c1 == c2 - 1 || c1 == c2 + 1) && r1 == r2 + 1))
        {
            //�������� ������-���������� � ��������-����������
            if (my_type == 1 && type_m == 2)
                FLAMEGPU->setVariable<double>("comfort_level", FLAMEGPU->getVariable<double>("comfort_level") - 0.1); // ��������� �������� ������ ������� �������� ��� �������� � ���������� 

            //�������� ������-�������� � ��������-����������
            if (my_type == 2 && type_m == 1)
                FLAMEGPU->setVariable<int>("language_level", FLAMEGPU->getVariable<int>("language_level") + 1); // ��������� ������ �������� �������
        }
    }

    return  flamegpu::ALIVE;
}


FLAMEGPU_AGENT_FUNCTION(looking_for_partner, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {


    unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    int my_state = FLAMEGPU->getVariable<int>("current_state");
    int my_gender = FLAMEGPU->getVariable<int>("gender");
    int my_type = FLAMEGPU->getVariable<float>("type");
    int my_married = FLAMEGPU->getVariable<int>("married");

    double max_distance = 100000000;
    int flag = 0;
    int ID_partner = 0;

    for (const auto& message : FLAMEGPU->message_in.wrap(my_x, my_y, 10)) // 10 ����� ������ ������ (49 - ����. ��� ����������� 100x100)
    {
        int ID = message.getVariable<int>("id");
        float x_m = message.getVariable<float>("x");
        float y_m = message.getVariable<float>("y");
        int state = message.getVariable<int>("current_state");
        int gender = message.getVariable<float>("gender");
        int type = message.getVariable<float>("type");
        int married = message.getVariable<int>("married");


        double distance = sqrt(pow(x_m - FLAMEGPU->getVariable<float>("x"), 2) + pow(y_m - message.getVariable<float>("y"), 2)); // ���������� �� ������ ��������

        if (my_gender != gender &&
            my_married == 0 && married == 0 &&
            my_state == 3 && state == 3 && distance < max_distance) // ���� ��� ������ ��������� � ��������� ������ ��������
        {
            flag = 1;
            ID_partner = ID;
            max_distance = distance;
        }
    }

    if (flag == 1) // ������� ������ � ������ ���� ����������� �������� �����
    {
        FLAMEGPU->setVariable<int>("married", 1);
        FLAMEGPU->setVariable<int>("ID_partner", ID_partner);
        // �������� ��������� ������-�������� ��� ������������ �������� (������) �����
        FLAMEGPU->message_out.setVariable<int>("ID_partner", FLAMEGPU->getVariable<int>("id")); // ����������� ID
        FLAMEGPU->message_out.setVariable<int>("ID_married", ID_partner); // ID ������ � ������� ����������� ���� 


        FLAMEGPU->setVariable<int>("move", 0); // ������� ������ � ������������ ���������
        FLAMEGPU->setVariable<int>("current_state", 1); // ������� ������ � ������������ ���������

    }

    if (flag == 0) // ������� �� ������
    {
        FLAMEGPU->message_out.setVariable<int>("ID_partner", 0); // ����������� ID
        FLAMEGPU->message_out.setVariable<int>("ID_married", 0); // ID ������ � ������� ����������� ���� 
        FLAMEGPU->setVariable<int>("move", 1); // ����������� ������ � ������������
    }


    FLAMEGPU->message_out.setIndex(my_x, my_y);

    return  flamegpu::ALIVE;
}


FLAMEGPU_AGENT_FUNCTION(looking_for_resource, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {

    unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0); // ���������� �������
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    int type_resource = FLAMEGPU->getVariable<int>("type_resource"); // ��� �������
    int is_occupied = FLAMEGPU->getVariable<int>("is_occupied");// ����� �� ������

    double max_distance = 100000000;
    int flag = 0;
    int ID_partner = 0;
    int r_res = 0;
    int c_res = 0;
    float x_res = 0;
    float y_res = 0;
    int ID_resource = 0;
    int ID_agent = 0;

    FLAMEGPU->setVariable("waiting_occupation", 0);

    int S_min = 10; // ����������� ����������� ������� ������ �������� �����
    float educ = FLAMEGPU->environment.getProperty<float>("Influence_of_education_on_employment_opportunities");

    for (const auto& message : FLAMEGPU->message_in(my_x, my_y, (dim/4)-1)) // 49 - ����. ��� ����������� 100x100
    {
        float x_m = message.getVariable<float>("x"); // ���������� ������
        float y_m = message.getVariable<float>("y");

        int my_state = message.getVariable<int>("current_state"); // ��������� ������
        int my_type = message.getVariable<int>("type"); // ��� ������

        double distance = sqrt(pow(x_m - FLAMEGPU->getVariable<float>("x"), 2) + pow(y_m - FLAMEGPU->getVariable<float>("y"), 2)); // ���������� �� ������

        max_distance = 10 * side_size * (1 + message.getVariable<double>("education_level") * educ);

      
        if (my_state == 2 && is_occupied == 0 && // ���� ����� ��������� � ��������� ������ �������� �����
            ((my_type == 1 && type_resource == 2) || (my_type == 2 && type_resource == 1)) && distance < max_distance && is_occupied == 0)
        {
            ID_agent = message.getVariable<int>("id"); // ID ������
            ID_resource = FLAMEGPU->getVariable<int>("id"); // ID �������
            x_res = FLAMEGPU->getVariable<float>("x");
            y_res = FLAMEGPU->getVariable<float>("y");
            r_res = FLAMEGPU->getVariable<int>("row");
            c_res = FLAMEGPU->getVariable<int>("clm");
            FLAMEGPU->setVariable<int>("waiting_occupation", 1); // �������� ������� ������� �����-���� �������
            FLAMEGPU->setVariable<int>("is_occupied", 1);
            flag = 1;
            max_distance = distance;

            FLAMEGPU->message_out.setVariable<float>("target_x", x_res);
            FLAMEGPU->message_out.setVariable<float>("target_y", y_res);
            FLAMEGPU->message_out.setVariable<int>("target_row", r_res);
            FLAMEGPU->message_out.setVariable<int>("target_clm", c_res);

        }
    }


    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<int>("row", FLAMEGPU->getVariable<int>("row"));
    FLAMEGPU->message_out.setVariable<int>("clm", FLAMEGPU->getVariable<int>("clm"));

    FLAMEGPU->message_out.setVariable<int>("agent_ID", ID_agent); // ID ������
    FLAMEGPU->message_out.setVariable<int>("target_ID", ID_resource); // ID �������, ������� ������ ���� �����

    FLAMEGPU->message_out.setIndex(my_x, my_y);

    return flamegpu::ALIVE;
}


FLAMEGPU_AGENT_FUNCTION(moving_trasaction, flamegpu::MessageArray2D, flamegpu::MessageNone)
{

    unsigned int my_x = FLAMEGPU->getVariable<unsigned  int, 2>("pos", 0);
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);
    int my_ID = FLAMEGPU->getVariable<int>("id");

    for (const auto& message : FLAMEGPU->message_in.wrap(my_x, my_y, 49)) // 49 ������������ ������ ��������� ��� ����������� 100 x 100
    {
        if (my_ID == message.getVariable<int>("agent_ID") && message.getVariable<int>("target_ID") != 0) // ���� ������� �� ������� id_�������
        {
            //   printf("%i, %i, moving_trasaction is run \n", message.getVariable<int>("target_ID"));

            float x_res = message.getVariable<float>("target_x");  // ���������� �������� �������
            float y_res = message.getVariable<float>("target_y");
            int r_res = message.getVariable<int>("target_row");
            int c_res = message.getVariable<int>("target_clm");

            FLAMEGPU->setVariable<float>("x", x_res);
            FLAMEGPU->setVariable<float>("y", y_res);

            FLAMEGPU->setVariable<int>("row", r_res);
            FLAMEGPU->setVariable<int>("clm", c_res);

            FLAMEGPU->setVariable<unsigned int, 2>("pos", 0, r_res);
            FLAMEGPU->setVariable<unsigned int, 2>("pos", 1, c_res);

            FLAMEGPU->setVariable<int>("move", 0); // ������� ������ � ������������ ���������
            FLAMEGPU->setVariable<int>("current_state", 1); // ������� ������ � ������������ ���������
        }
    }

    return flamegpu::ALIVE;

}



FLAMEGPU_AGENT_FUNCTION(get_married, flamegpu::MessageArray2D, flamegpu::MessageNone) {

    unsigned int my_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int my_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    int my_ID = FLAMEGPU->getVariable<int>("id");
    for (const auto& message : FLAMEGPU->message_in(my_x, my_y, dim))
    {
        int ID_partner = message.getVariable<int>("ID_partner");
        int ID_married = message.getVariable<int>("ID_married");

        if (my_ID == ID_married)
        {
            FLAMEGPU->setVariable<int>("ID_partner", ID_partner); // ����� � ��������� �����������
            FLAMEGPU->setVariable<int>("married", 1); // ����� � ��������� �����������
        }
    }
    return flamegpu::ALIVE;
}




FLAMEGPU_AGENT_FUNCTION(update_agent_state, flamegpu::MessageNone, flamegpu::MessageNone) {


    int current_age = FLAMEGPU->getVariable<int>("age") + FLAMEGPU->getStepCounter() - FLAMEGPU->getVariable<int>("t_arrival");
    double comfort_level = FLAMEGPU->getVariable<double>("comfort_level");
    int type = FLAMEGPU->getVariable<int>("type");
    int married = FLAMEGPU->getVariable<int>("married");
    int kids = FLAMEGPU->getVariable<int>("kids");
    int previous_state = FLAMEGPU->getVariable<int>("previous_state");
    int unemployed = FLAMEGPU->getVariable<int>("unemployed");
    int type_resource = FLAMEGPU->getVariable<int>("type_resource");
    int t_arrival = FLAMEGPU->getVariable<int>("t_arrival");
    int language_level = FLAMEGPU->getVariable<int>("language_level");
    double education_level = FLAMEGPU->getVariable<double>("education_level");
    double Time_for_assimilation = FLAMEGPU->getVariable<double>("Time_for_assimilation");

    int max_natives_age = FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_natives");
    int max_migrants_age = FLAMEGPU->environment.getProperty<uint8_t>("Average_life_time_of_migrants");
    int min_natives_age = FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_natives");
    int min_migrants_age = FLAMEGPU->environment.getProperty<uint8_t>("Age_for_married_and_kids_birth_of_migrants");
    int min_natives_comfort = FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_natives");
    int min_migrants_comfort = FLAMEGPU->environment.getProperty<uint8_t>("Minimum_comfort_level_of_migrants");
    int pension_age = FLAMEGPU->environment.getProperty<uint8_t>("Pension_age");


    FLAMEGPU->setVariable<int>("kids_should_be_born_native", 0);  // �� ��������� ����� ����� ���
    FLAMEGPU->setVariable<int>("kids_should_be_born_migrant", 0);  

    if (type == 1)
        FLAMEGPU->setVariable<double>("Time_for_assimilation", 0.0);

    double t_assim = 30; // ������������ ���� �� �����������

    double koeff = FLAMEGPU->environment.getProperty<float>("Average_GDP_rate") * (FLAMEGPU->environment.getProperty<double>("Expenditure_on_education_share"));
    if (koeff <= 0)
        koeff = 1;

    education_level = 3 * exp(-1 / (double)(koeff * (1 + language_level))); // ������� �����������

    if (language_level > 0 && education_level > 0)
        t_assim = 30 * pow(1 / (double)(1 + language_level), 0.7) * pow(1 / (double)(1 + education_level), 0.3); //����� ������ �����������

    if (t_assim > 30)
        t_assim = 30;
    if (t_assim < 0)
        t_assim = 0;

    if (type == 2)
        FLAMEGPU->setVariable<double>("Time_for_assimilation", t_assim);


    //��������� ������� ������� �������� ������
    if (comfort_level > 0 && comfort_level < 10 && type == 1) // ����������
    {
        if (type_resource == 2) //������������������� ������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level + 1);
        if (type_resource == 1) //������������������ ������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level + 0.5);
        if (type_resource == 0) // �����������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level - 1);
    }

    if (comfort_level > 0 && comfort_level < 10 && type == 2) // ��������
    {
        if (type_resource == 2) //������������������� ������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level + 1);
        if (type_resource == 1) //������������������ ������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level + 1);
        if (type_resource == 0) // �����������
            FLAMEGPU->setVariable<double>("comfort_level", comfort_level - 1);
    }

    // ������������� ��������� ������
    if ((previous_state == 2 && comfort_level >= min_natives_comfort) ||
        ((previous_state == 3 || previous_state == 4 || previous_state == 5) && current_age > 50) && type == 1 ||

        (previous_state == 2 && comfort_level >= min_migrants_comfort) ||
        ((previous_state == 3 || previous_state == 4 || previous_state == 5) && current_age > 50) && type == 2)
        FLAMEGPU->setVariable<int>("current_state", 1);  // ������������   


 // ����� ������
    if ((comfort_level < min_natives_comfort &&
        current_age >= 18 &&
        current_age <= pension_age && unemployed == 1 && type == 1 && (previous_state == 1 || previous_state == 4 || previous_state == 5)) ||

        (comfort_level < min_migrants_comfort &&
            current_age >= 18 &&
            current_age <= pension_age && unemployed == 1 && type == 2 && (previous_state == 1 || previous_state == 4 || previous_state == 5)))
        FLAMEGPU->setVariable<int>("current_state", 2);  // ����� ������   



// ����� �������� ��� ���������� � ����
    if ((comfort_level >= min_natives_comfort &&
        current_age >= min_natives_age &&
        current_age <= 50 && married == 0 && previous_state == 1 && type == 1) ||

        (comfort_level >= min_migrants_comfort &&
            current_age >= min_migrants_age &&
            current_age <= 50 && married == 0 && previous_state == 1 && type == 2))
        FLAMEGPU->setVariable<int>("current_state", 3); // ����� �������� ��� ����� � �������� �����     


   //���������� � �������� �����
    if ((comfort_level >= min_natives_comfort &&
        current_age >= min_natives_age &&
        current_age <= 50 && married == 1 && type == 1 && (previous_state == 3 || previous_state == 1)) ||

        (comfort_level >= min_migrants_comfort &&
            current_age >= min_migrants_age &&
            current_age <= 50 && married == 1 && type == 2 && (previous_state == 3 || previous_state == 1)))
        FLAMEGPU->setVariable<int>("current_state", 4); // ����������� � �������� �����    


    double p = FLAMEGPU->random.uniform<double>();
	double g = -1 / (pow((double)kids, 0.9) * pow((double)current_age, 0.1));
	double w = (double)exp(g);
    
    if ((previous_state == 4 && p < 1 * (1 - w)) ||
        (previous_state == 1 && married == 0 && p < 0.01 * (1 - w)))
    {
        FLAMEGPU->setVariable<int>("current_state", 5); // �������� ������   
        if (type == 1)
            FLAMEGPU->setVariable<int>("kids_should_be_born_native", 1);
        if (type == 2)
            FLAMEGPU->setVariable<int>("kids_should_be_born_migrant", 1);


                
        FLAMEGPU->setVariable<int>("kids", FLAMEGPU->getVariable<int>("kids") + 1); // �������� ������   
    }


    //����������� ���������
    if (FLAMEGPU->getStepCounter() - t_arrival > t_assim && type == 2)
    {
        //  printf("%f, %i, %f\n", t_assim, language_level, education_level);

          //���������� ������-�������� � ����� ��������� �������-�������
        FLAMEGPU->setVariable<int>("current_state", 6);
        //���������� ������-�������� � ����� ��������� �������-�������
        FLAMEGPU->setVariable<int>("type", 1);
        FLAMEGPU->setVariable<int>("is_assimilated", 1); // ����� � ���, ��� ��� ������ ���������������� �������
    }


    FLAMEGPU->setVariable<int>("previous_state", FLAMEGPU->getVariable<int>("current_state"));

    if (current_age >= FLAMEGPU->environment.getProperty<uint8_t>("Pension_age"))
        FLAMEGPU->setVariable<int>("is_pensioner", 1);



    //������� ������� �� ������������ �������� 
    if ((current_age <= max_natives_age && FLAMEGPU->getVariable<int>("type") == 1) ||
        (current_age <= max_migrants_age && FLAMEGPU->getVariable<int>("type") == 2))
        return flamegpu::ALIVE;
    else
        return flamegpu::DEAD;
}


int main(int argc, const char** argv) {

    flamegpu::ModelDescription model("Migration");

    flamegpu::MessageArray2D::Description& message_resource = model.newMessage<flamegpu::MessageArray2D>("location_resource");
    {
        message_resource.newVariable<int>("id");
        message_resource.newVariable<float>("x");
        message_resource.newVariable<float>("y");

        message_resource.newVariable<int>("type_resource"); // �������� ���� ����� ��� ���������� ��������� �������
        message_resource.newVariable<int>("is_occupied"); // �������� ��������� �����
        message_resource.newVariable<int>("row"); // ���������� ������ � ���������� ������� ���������
        message_resource.newVariable<int>("clm");
        message_resource.newVariable<int>("request_agent");

        message_resource.setDimensions(dim, dim);
    }


    flamegpu::MessageArray2D::Description& message_agent = model.newMessage<flamegpu::MessageArray2D>("location_agent");
    {
        message_agent.newVariable<float>("x");
        message_agent.newVariable<float>("y");

        message_agent.newVariable<int>("id");
        message_agent.newVariable<int>("row"); // ���������� ������ � ���������� ������� ���������
        message_agent.newVariable<int>("clm");

        message_agent.newVariable<int>("move");


        message_agent.newVariable<int>("current_state");
        message_agent.newVariable<int>("type");
        message_agent.newVariable<int>("married");
        message_agent.newVariable<int>("gender");

        message_agent.newVariable<double>("education_level");

        message_agent.setDimensions(dim, dim);
    }


    flamegpu::MessageArray2D::Description& message_married = model.newMessage<flamegpu::MessageArray2D>("agents_married");
    {
        message_married.newVariable<int>("ID_partner");
        message_married.newVariable<int>("ID_married");

        message_married.setDimensions(dim, dim);
    }


    flamegpu::MessageArray2D::Description& message_occupied = model.newMessage<flamegpu::MessageArray2D>("resource_occupied");
    {

        message_occupied.newVariable<int>("id");// ������� ���������� �������
        message_occupied.newVariable<float>("x");
        message_occupied.newVariable<float>("y");
        message_occupied.newVariable<int>("row");
        message_occupied.newVariable<int>("clm");

        message_occupied.newVariable<float>("target_x");// ���������� �������� �������
        message_occupied.newVariable<float>("target_y");
        message_occupied.newVariable<int>("target_row");
        message_occupied.newVariable<int>("target_clm");

        message_occupied.newVariable<int>("agent_ID"); // 
        message_occupied.newVariable<int>("target_ID"); // 

        message_occupied.setDimensions(dim, dim);
    }

    flamegpu::AgentDescription& agent = model.newAgent("agent");
    {
        agent.newVariable<int>("id"); // ID
        agent.newVariable<float>("x"); // ���������� ������ � ���������� ������� ���������
        agent.newVariable<float>("y");
        agent.newVariable<int>("type"); // 1 - �������� ������, 2 - �������� 
        agent.newVariable<int>("row"); // ���������� ������ � ���������� ������� ���������
        agent.newVariable<int>("clm");
        agent.newVariable <unsigned int, 2>("pos"); // ������� ������ � ��������� ������������
        agent.newVariable<int>("current_state");
        agent.newVariable<int>("previous_state");
        agent.newVariable<int>("unemployed");
        agent.newVariable<int>("t_arrival");
        agent.newVariable<int>("age");
        agent.newVariable<int>("gender");
        agent.newVariable<double>("education_level");
        agent.newVariable<int>("language_level");
        agent.newVariable<double>("comfort_level");
        agent.newVariable<int>("married");
        agent.newVariable<int>("kids");
        agent.newVariable<int>("kids_should_be_born_native");
        agent.newVariable<int>("kids_should_be_born_migrant");
        agent.newVariable<double>("Time_for_assimilation");
        agent.newVariable<int>("ID_partner");
        agent.newVariable<int>("type_resource"); // ��� ������� - �������� �����, ������� �������� �����
        agent.newVariable<int>("move"); // ������� ����, ��� ����� ������ ������������ � ������������ ����� ����������� ������������
      //  agent.newVariable<int>("CI");
        agent.newVariable<int>("is_pensioner"); // �������� �� �����������
        agent.newVariable<int>("is_assimilated"); // �������� �� ���������������� ���������
    }



    auto& fn_all_agents = agent.newFunction("all_agents", all_agents);
    {
        fn_all_agents.setMessageOutput("location_agent"); // ���������� �� ���� �������
    }

   
    auto& fn_check_all_agents = agent.newFunction("check_all_agents", check_all_agents);
    {
        fn_check_all_agents.setMessageInput("location_agent"); 
        fn_check_all_agents.setAllowAgentDeath(true); // �������� ������ �������� ������������ � ��� �� ������
    }


    auto& fn_neighbour_agents1 = agent.newFunction("agent_to_agent_contacts", agent_to_agent_contacts);
    {
        fn_neighbour_agents1.setMessageInput("location_agent");
    }


    auto& fn_agent_location_update = agent.newFunction("check_cell", check_cell);
    {
        fn_agent_location_update.setMessageInput("location_resource");
       // fn_agent_location_update.setMessageOutput("location_agent");
    }


    auto& fn_state_update = agent.newFunction("update_agent_state", update_agent_state);  // ���������� ��������� ������

    {
        fn_state_update.setAllowAgentDeath(true);
    }

    auto& fn_looking_for_partner = agent.newFunction("looking_for_partner", looking_for_partner); // ��������� ������, ����������� �� ������������� ���������
    {
        fn_looking_for_partner.setMessageInput("location_agent");
        fn_looking_for_partner.setMessageOutput("agents_married");
    }


    auto& fn_married = agent.newFunction("get_married", get_married); // �������� ������ �� ������� � ����� �������������� � �������
    {
        fn_married.setMessageInput("agents_married");
    }

    auto& fn_moving_trasaction = agent.newFunction("moving_trasaction", moving_trasaction); // ��������� ������, ����������� �� ������������� ���������
    {
        fn_moving_trasaction.setMessageInput("resource_occupied");
    }

    flamegpu::AgentDescription& resources = model.newAgent("resources");
    {
        resources.newVariable<int>("id"); // ID
        resources.newVariable<float>("x"); // ���������� ������ � ���������� ������� ���������
        resources.newVariable<float>("y");
        resources.newVariable<int>("type_resource"); // 2 - �������������������, 1 - ������������������, 0 - �����������
        resources.newVariable<int>("row"); // ���������� ������ � ���������� ������� ���������
        resources.newVariable<int>("clm");
        resources.newVariable <unsigned int, 2>("pos"); // ������� ������� � ��������� ������������
        resources.newVariable<int>("time_creation"); // ������ ������� �������� �������
        resources.newVariable<int>("is_occupied"); // ������ �����
        resources.newVariable<int>("waiting_occupation"); // ������ ������� ������� �������
        resources.newVariable<int>("request_agent"); // ID ������, ������� ����� ������������� � ������ ������
        //resources.newVariable<int>("CI"); // ������ ������������� ������ (�� ��������� - 1)
    }


    auto& fn_all_resources = resources.newFunction("all_resources", all_resources);
    {
        fn_all_resources.setMessageOutput("location_resource"); // ���������� �� ���� ��������
    }


    auto& fn_resources_location_update = resources.newFunction("update_cell", update_cell);
    {
        fn_resources_location_update.setMessageInput("location_agent");
       // fn_resources_location_update.setMessageOutput("location_resource");
    }

    auto& fn_workplaces_creation = resources.newFunction("workplaces_creation", workplaces_creation); // �������� ����� ������� ���
    {
        fn_workplaces_creation.setMessageInput("location_resource");
    }


    auto& fn_looking_for_resource = resources.newFunction("looking_for_resource", looking_for_resource); // "������ ���� ������"
    {
        fn_looking_for_resource.setMessageInput("location_agent");
        fn_looking_for_resource.setMessageOutput("resource_occupied"); // ID �������, ������� ������ ������ ������������ �����
    }


    /**
    * Control flow
    */

    model.addInitFunction(init_function);
    model.addStepFunction(cells_update);
    model.addStepFunction(BasicOutput);
    model.addExitCondition(exit_condition);
       

    { // Layer  #0 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(all_agents);
    }

    { // Layer  #1 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(all_resources);
    }

    

    {   // Layer #2 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(check_all_agents);
    }

    

    {   // Layer #3 ������� ������
       flamegpu::LayerDescription& layer = model.newLayer();
       layer.addAgentFunction(check_cell);
    }

    
    {   // Layer #4 ������� �������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(update_cell);
    }

  
    
    {   // Layer #5 ������� �������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(workplaces_creation);
    }

    
    {   // Layer #6 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(update_agent_state);
    }


    {   // Layer #7 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(agent_to_agent_contacts);
    }


    {   // Layer #8 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(looking_for_partner);
    }


    {   // Layer #9 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(get_married);
    }


    {   // Layer #10 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(looking_for_resource);
    }


    {   // Layer #11 ������� ������
        flamegpu::LayerDescription& layer = model.newLayer();
        layer.addAgentFunction(moving_trasaction);
    }
    
    {
        flamegpu::EnvironmentDescription& env = model.Environment();
        //����������� ��������� ������
        env.newProperty<uint8_t>("Life_age_high_technology_work_places", 15);
        env.newProperty<uint8_t>("Life_age_low_technology_work_places", 15);
        env.newProperty<uint8_t>("Method_work_places_creation", 1);
        env.newProperty<uint8_t>("Frequency_work_places_creation", 5);
        env.newProperty<uint8_t>("Average_life_time_of_natives", 80);
        env.newProperty<uint8_t>("Average_life_time_of_migrants", 65);

        env.newProperty<double>("Share_of_new_migrants", 0.1);
        env.newProperty<double>("Expenditure_on_education_share", 0.1); // ���� ������ �� ����������� � ��� �� ���� ���������


        env.newProperty<uint8_t>("Age_for_married_and_kids_birth_of_natives", 29);
        env.newProperty<uint8_t>("Age_for_married_and_kids_birth_of_migrants", 18);
        env.newProperty<uint8_t>("Minimum_comfort_level_of_natives", 6);
        env.newProperty<uint8_t>("Minimum_comfort_level_of_migrants", 3);
        env.newProperty<uint8_t>("Pension_age", 65);

        env.newProperty<float>("Influence_of_education_on_employment_opportunities", 0.5);

        //������������������ ����������
        env.newProperty<float>("GE", 0);
        env.newProperty<float>("GDP", 0);

        env.newProperty<float>("GE_rate_total", 0);
        env.newProperty<float>("GDP_rate_total", 0);
        
        env.newProperty<float>("Average_Government_Expenditure_rate", 0);
        env.newProperty<float>("Average_GDP_rate", 0);

        env.newProperty<float>("Total_DSI", 0);
        env.newProperty<float>("Average_DSI", 0);

        env.newProperty<int, 2>("Center_of_first_high_tech_cluster", { 0, 0 }); // ���������� ������� ��������� ��� �������� ������� ����
        env.newProperty<int, 2>("Center_of_second_high_tech_cluster", { 0, 0 });
        env.newProperty<int, 2>("Center_of_third_high_tech_cluster", { 0, 0 });

        env.newProperty<int, 2>("Center_of_first_low_tech_cluster", { 0, 0 });
        env.newProperty<int, 2>("Center_of_second_low_tech_cluster", { 0, 0 });
        env.newProperty<int, 2>("Center_of_third_low_tech_cluster", { 0, 0 });

        env.newProperty<int>("Size_of_high_tech_cluster", 10);
        env.newProperty<int>("Size_of_low_tech_cluster", 10);
        
    }

    /**
     * Create Model Runner
     */

    flamegpu::CUDASimulation cuda_model(model);
    cuda_model.initialise(1, argv);
    cuda_model.SimulationConfig().steps = TIME_STOP;
    flamegpu::AgentVector population1(model.Agent("agent"), NumberOfCitizens + NumberOfMigrants); 
    flamegpu::AgentVector population2(model.Agent("resources"), dim*dim);



    //������������� ������ ��� ������������
    if(VIS_MODE==0)
    { 
        flamegpu::RunPlanVector runs(model, RUN_COUNT); // 2 - ���������� ��������
        {
            runs.setSteps(TIME_STOP);

            runs.setRandomSimulationSeed(12, 1);

            runs.setOutputSubdirectory("results");

            runs.setPropertyUniformRandom<double>("Share_of_new_migrants", double(0.1f), double(0.5f));
            runs.setPropertyUniformRandom<double>("Expenditure_on_education_share", double(0.1f), double(0.5f));

            runs.setPropertyUniformDistribution<uint8_t>("Life_age_high_technology_work_places", 5, 15);
            runs.setPropertyUniformDistribution<uint8_t>("Life_age_low_technology_work_places", 5, 15);
            runs.setPropertyUniformDistribution<uint8_t>("Method_work_places_creation", 1, 2);
            runs.setPropertyUniformDistribution<uint8_t>("Frequency_work_places_creation", 5, 5);
            runs.setPropertyUniformDistribution<uint8_t>("Average_life_time_of_natives", 70, 90);
            runs.setPropertyUniformDistribution<uint8_t>("Average_life_time_of_migrants", 60, 80);

            runs.setPropertyUniformDistribution<uint8_t>("Age_for_married_and_kids_birth_of_natives", 20, 35);
            runs.setPropertyUniformDistribution<uint8_t>("Age_for_married_and_kids_birth_of_migrants", 18, 30);
            runs.setPropertyUniformDistribution<uint8_t>("Minimum_comfort_level_of_natives", 6, 10);
            runs.setPropertyUniformDistribution<uint8_t>("Minimum_comfort_level_of_migrants", 3, 6);
            runs.setPropertyUniformDistribution<uint8_t>("Pension_age", 60, 75);

        }

        flamegpu::CUDAEnsemble cuda_ensemble(model, 1, argv);
        cuda_ensemble.simulate(runs);
    }
 
    //��������� ������ � �������������
    if (VIS_MODE == 1)
    {
        initVisualisation();
        glutTimerFunc(1, timer, 0);
        cuda_model.SimulationConfig().steps = TIME_STOP;
        std::thread first([&cuda_model]() { cuda_model.simulate(); });
        runVisualisation(); //������ ������������
        first.join();
    }
    
     

    // �������� ���������� ����������� (������ ��� ��������)

   for (int i = 0; i < RUN_COUNT; i++)
   {
       if (out.is_open())
	   {
           if (i == 0)
           {
               out << "Share_of_new_migrants" <<
                   ";" << "Expenditure_on_education_share" <<
                   ";" << "Life_age_high_technology_work_places" <<
                   ";" << "Life_age_low_technology_work_places" <<
                   ";" << "Frequency_work_places_creation" <<
                   ";" << "Average_life_time_of_natives" <<
                   ";" << "Average_life_time_of_migrants" <<
                   ";" << "Age_for_married_and_kids_birth_of_natives" <<
                   ";" << "Age_for_married_and_kids_birth_of_migrants" <<
                   ";" << "Minimum_comfort_level_of_natives" <<
                   ";" << "Minimum_comfort_level_of_migrants" <<
                   ";" << "Pension_age" <<
                   ";" << "Method_work_places_creation" <<

                   ";" << "Total_count_of_agents" <<
                   ";" << "Share_of_non-assimilated_migrants" <<
                   ";" << "Number_of_assimilated_migrants" <<
                   ";" << "Averaged_time_for_assimilation" <<
                   ";" << "Duncan_Segregation_Index" <<
                   ";" << "Average_GDP_rate" <<
                   ";" << "Average_Government_Expenditure_rate" << std::endl;
           }



		   //�������� ���������� �����������
           
           float par1 = (float)myArray1[i].load();
           float par2 = (float)myArray2[i].load();
           float par3 = (float)myArray15[i].load();
           float par4 = (float)myArray17[i].load();
           float par5 = (float)myArray18[i].load();
           float par6 = (float)myArray19[i].load();
           float par7 = (float)myArray20[i].load();

             out << par1 / 100000 <<
               ";" << par2 / 100000 <<
               ";" << myArray3[i].load() <<
               ";" << myArray4[i].load() <<
               ";" << myArray5[i].load() <<
               ";" << myArray6[i].load() <<
               ";" << myArray7[i].load() <<
               ";" << myArray8[i].load() <<
               ";" << myArray9[i].load() <<
               ";" << myArray10[i].load() <<
               ";" << myArray11[i].load() <<
               ";" << myArray12[i].load() <<
               ";" << myArray13[i].load() <<

               ";" << myArray14[i].load() <<
               ";" << par3 / 100000 <<
			   ";" << myArray16[i].load() <<
			   ";" << par4 / 100000 <<
			   ";" << par5 / 100000 <<
               ";" << par6 / 100000 <<
			   ";" << par7 / 100000 << std::endl;
	   }

   }

    out.close();


    //����� ����� ���������� ������ ������ �� ������ ������� (��� ��������)
   
    for (int i = 0; i < RUN_COUNT; i++)
    { 
        if (out2.is_open())
        {
            //�������� ���������� �����������
            out2 << myArray_id[i] <<
             ";" << myArray_test1[i] <<
             ";" << myArray_test2[i] << 
             ";" << myArray_test3[i] <<  std::endl;
        }
    }

    out2.close();
    

    return 0;
}
